#include "hip/hip_runtime.h"


#include <stdio.h>
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"




#define N 100 /* runners*/

#define MAX 6 /* runners max speed 5 */

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

    /* we have to initialize the state */
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, unsigned int* speeds) {


    speeds[blockIdx.x] = hiprand(&states[blockIdx.x]) % 6;
    if (speeds[blockIdx.x] == 0)
        speeds[blockIdx.x] += 1;

}


//variables



int main() {

    hiprandState_t* states;
    unsigned int cpu_nums[N];
    unsigned int* gpu_nums;
    unsigned int location[N];
    unsigned int wait = 1000000000;
    unsigned int i;

    
        /* allocate space on the GPU for the random states */
        hipMalloc((void**)&states, N * sizeof(hiprandState_t));

        /* invoke the GPU to initialize all of the random states */
        init << < N, 1 >> > (time(0), states);

        /* allocate an array of unsigned ints on the CPU and GPU */

        hipMalloc((void**)&gpu_nums, N * sizeof(unsigned int));

        /*  kernel to get some random numbers */
        randoms << < N, 1 >> > (states, gpu_nums);

        /* copy the random numbers back */
        hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

        /* print them out */
        for ( i = 0; i < N; i++) {
            location[i] = cpu_nums[i] + cpu_nums[i];
            
            
               
            printf("%d  nolu yarismaci \t %u anlik hizi \t %u yarismaci konumu \n", i, cpu_nums[i], location[i]);
         }

        /* free the memory we allocated for the states and numbers */
        hipFree(states);
        hipFree(gpu_nums);
        

    

    
   return 0;
}